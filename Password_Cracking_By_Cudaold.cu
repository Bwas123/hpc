#include "hip/hip_runtime.h"
// to compile
// nvcc -o pc Password_Cracking_By_Cudaold.cu  -lm -lcrypt

#include <hip/hip_runtime_api.h>
#include <stdio.h>

__device__ int is_a_match(char *check) {	// Compares each password attempt.
  char password[] = "BI1245";

  char *b = check;
  char *c = password;

  while(*b == *c) {
    if(*b == '\0') {
      return 1;
    }
    b++;
    c++;
  }
  return 0; //returns 0 
}

__global__ void kernel() {
  //char *alphabet = "ABCDEFGHIJKLMNOPQRSTUVWXYZ";
  char alpha[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
  //alphabet[27] = '\0';
  char num[10] = {'0','1','2','3','4','5','6','7','8','9'};
  //numbers[11] = '\0';
  char check[7];
  check[6] = '\0';
  int i, j, k, l;
  for(i=0;i<10;i++) {
     for(j=0; j<10; j++) {
	for(k=0;k<10;k++) {
     	   for(l=0; l<10; l++) {
          //printf("%c%c\n", alphabet[blockIdx.x], alphabet[threadIdx.x], numbers[i], numbers[j]);
          check[0] = alpha[blockIdx.x];
          check[1] = alpha[threadIdx.x];
          check[2] = num[i];
          check[3] = num[j];
	  check[4] = num[k];
          check[5] = num[l];
          //printf("%s \n", test);

          if(is_a_match(check)) {
          printf("Password successfully cracked: %s\n", check);
          } //else {
               //printf("tried: %s\n", check);
          //}
     }
}}
  }
}

int time_difference(struct timespec *start, struct timespec *finish, long long int *difference)
 {
	  long long int ds =  finish->tv_sec - start->tv_sec; 
	  long long int dn =  finish->tv_nsec - start->tv_nsec; 

	  if(dn < 0 ) 
	  {
	    ds--;
	    dn += 1000000000; 
          } 

	  *difference = ds * 1000000000 + dn;
	  return !(*difference > 0);
}





	//Calculating time



int main(int argc, char *argv[])
{

	
  kernel <<<26, 26>>>();

  hipDeviceSynchronize();

  	
	struct timespec start, finish;   
  	long long int time_elapsed;

  	clock_gettime(CLOCK_MONOTONIC, &start);


	clock_gettime(CLOCK_MONOTONIC, &finish);
	  time_difference(&start, &finish, &time_elapsed);
	  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed,
		                                 (time_elapsed/1.0e9)); 
  return 0;
}


